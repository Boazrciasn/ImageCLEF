#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include <stdlib.h>
#include <opencv2/core/cuda.hpp>

#define N 512

__global__ void add(int *a, int *b, int *c, cv::cuda::PtrStepSz<uchar> asdf) {
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

void kernel_wrapper(int *a, int *b, int *c)
{
	int *d_a, *d_b, *d_c;
	int size = N*sizeof(int);

	// Alloc space for device copies of a, b, c
	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

    // Copy inputs to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	cv::cuda::GpuMat test(cv::Mat_<uchar>::zeros(2,2));

    // Launch add() kernel on GPU with N blocks
    add<<<N,1>>>(d_a, d_b, d_c,test);
//	gpuErrchk( hipPeekAtLastError() );
    // Copy result back to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);
    hipFree(a); hipFree(b); hipFree(c);
}


__global__ void img_proc_kernel(cv::cuda::PtrStepSz<uchar3> img,
								cv::cuda::PtrStepSz<float> layered_hist,
								cv::cuda::PtrStepSz<int> tree_nodes,
								cv::cuda::PtrStepSz<float>features,
								int lbl_count,
								int padding)
{
	int xIndex = blockIdx.x * blockDim.x + threadIdx.x + padding;
	int yIndex = blockIdx.y * blockDim.y + threadIdx.y + padding;

	int n = features.cols/2;
	int* current_node = tree_nodes.ptr(0);

	int leftID = current_node[0];
	int ftrID, tau, fr1_x, fr1_y, fr2_x, fr2_y;
	float* feature;

	while(leftID != -1)
	{
		ftrID   = current_node[1];
		feature = features.ptr(ftrID);
		tau     = current_node[2];
		fr1_x   = xIndex + current_node[3];
		fr1_y   = yIndex + current_node[4];
		fr2_x   = xIndex + current_node[5];
		fr2_y   = yIndex + current_node[6];

		uchar3 ftr1{0};
		uchar3 ftr2{0};

		if(ftrID == 0)
		{
			ftr1 = img(fr1_y, fr1_x);
			ftr2 = img(fr2_y, fr2_x);
		}
		else
		{
			for (int i = 0; i < 2 * n; ++i) {
				uchar3 pxVal = img(fr1_y + i / n, fr1_x + i % n);
				ftr1.x = (uchar) (pxVal.x * feature[i]);
				ftr1.y = (uchar) (pxVal.y * (uchar) feature[i]);
				ftr1.z = (uchar) (pxVal.z * (uchar) feature[i]);
			}
		}

		uchar a[3];
		a[0] = ftr1.x-ftr2.x;
		a[1] = ftr1.y-ftr2.y;
		a[2] = ftr1.z-ftr2.z;
		uchar val = a[0]*a[0] + a[1]*a[1] + a[2]*a[2];

		if(val <= tau*tau)
			current_node = tree_nodes.ptr(leftID);
		else
			current_node = tree_nodes.ptr(leftID+1);
		leftID     = current_node[0];
	}

	img(xIndex,yIndex).x = 255;
	img(xIndex,yIndex).y = 0;
	img(xIndex,yIndex).z = 0;
}

void img_proc(cv::Mat& img, cv::Mat_<float>& layered_hist, cv::Mat_<int>& tree_nodes, cv::Mat_<float>& features, int lbl_count, int padding)
{
	dim3 block_size(32,16); // 512 threads
	dim3 grid_size;
	grid_size.x = (img.cols + block_size.x - 1)/block_size.x;  // Greater than or equal to image width
	grid_size.y = (img.rows + block_size.y - 1)/block_size.y;  // Greater than or equal to image height

	cv::copyMakeBorder(img, img, padding, padding, padding, padding, cv::BORDER_CONSTANT);

	cv::cuda::GpuMat img_d(img);
	cv::cuda::GpuMat layered_hist_d(layered_hist);
	cv::cuda::GpuMat nodes_d(tree_nodes);
	cv::cuda::GpuMat features_d(features);

	img_proc_kernel<<<grid_size,block_size>>>(img_d, layered_hist_d, nodes_d, features_d, lbl_count, padding);
	img_d.download(img);
}































