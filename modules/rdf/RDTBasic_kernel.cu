#include "hip/hip_runtime.h"
#include <opencv2/core/mat.hpp>
#include <opencv2/core/cuda.hpp>
#include <opencv2/highgui.hpp>
#include <hip/hip_runtime.h>
#include <>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
    if (code != hipSuccess)
    {
        fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__device__ inline float norm_device(float3 in1, float3 in2)
{
    float a[3];
    a[0] = in1.x - in2.x;
    a[1] = in1.y - in2.y;
    a[2] = in1.z - in2.z;
    return (a[0]*a[0] + a[1]*a[1] + a[2]*a[2]);
}

__global__ void compute_layered_hist_gpu(cv::cuda::PtrStepSz<uchar3> img,
                                         cv::cuda::PtrStepSz<float> layered_hist,
                                         cv::cuda::PtrStepSz<int> tree_nodes,
                                         cv::cuda::PtrStepSz<float>features,
                                         int lbl_count,
                                         int padding_x,
                                         int padding_y)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x + padding_x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y + padding_y;

    if(xIndex >= (img.cols - padding_x) || yIndex >= (img.rows - padding_y))
        return;

    int n = (int)sqrtf((float)features.cols);  // N is one side of a squared feature
    int* current_node = tree_nodes.ptr(0);

    int rightID = current_node[0];
    int leftID  = 1;
    int ftrID, tau, fr1_x, fr1_y, fr2_x, fr2_y;
    float* feature;

    while(rightID != -1)
    {
        ftrID   = current_node[1];
        feature = features.ptr(ftrID);
        tau     = current_node[2];
        fr1_x   = xIndex + current_node[3];
        fr1_y   = yIndex + current_node[4];
        fr2_x   = xIndex + current_node[5];
        fr2_y   = yIndex + current_node[6];

        float3 ftr1{0};
        float3 ftr2{0};


        if(ftrID == 0)
        {
            uchar3 pxVal = img(fr1_y, fr1_x);
            ftr1.x = pxVal.x;
            ftr1.y = pxVal.y;
            ftr1.z = pxVal.z;

            pxVal = img(fr2_y, fr2_x);
            ftr2.x = pxVal.x;
            ftr2.y = pxVal.y;
            ftr2.z = pxVal.z;
        }
        else
            for(int i = 0; i < n*n; ++i)
            {
                uchar3 pxVal = img(fr1_y + i/n, fr1_x + i%n);
                ftr1.x += pxVal.x*feature[i];
                ftr1.y += pxVal.y*feature[i];
                ftr1.z += pxVal.z*feature[i];
            }

        if(norm_device(ftr1,ftr2) <= tau*tau) {
            current_node = tree_nodes.ptr(leftID);
            leftID++;
        }
        else {
            current_node = tree_nodes.ptr(rightID);
            leftID = rightID + 1;
        }
        rightID = current_node[0];
    }

    for (int i = 0; i < lbl_count; ++i)
        layered_hist(yIndex - padding_y, (xIndex - padding_x)*lbl_count + i) = current_node[i+1]/100.0f;
}






////////////////////////////////////////////////////////////////////////////////////
/// \brief compute_layered_hist_gpu
/// \param img
/// \param layered_hist
/// \param tree_nodes
/// \param features
/// \param lbl_count
/// \param padding_x
/// \param padding_y
///////////////////////////////////////////////////////////////////////////////////////

__global__ void compute_layered_hist_gpu_2b(cv::cuda::PtrStepSz<uchar> img,
                                            cv::cuda::PtrStepSz<float> layered_hist,
                                            cv::cuda::PtrStepSz<int> tree_nodes,
                                            cv::cuda::PtrStepSz<float>features,
                                            int lbl_count,
                                            int padding_x,
                                            int padding_y)
{
    int xIndex = blockIdx.x * blockDim.x + threadIdx.x + padding_x;
    int yIndex = blockIdx.y * blockDim.y + threadIdx.y + padding_y;

    if(xIndex >= (img.cols - padding_x) || yIndex >= (img.rows - padding_y))
        return;
    if(img(yIndex, xIndex) == 0)
        return;

    int n = (int)sqrtf((float)features.cols);  // N is one side of a squared feature
    int* current_node = tree_nodes.ptr(0);

    int rightID = current_node[0];
    int leftID  = 1;
    int ftrID, tau, fr1_x, fr1_y, fr2_x, fr2_y;
    float* feature;

    while(rightID != -1)
    {
        ftrID   = current_node[1];
        feature = features.ptr(ftrID);
        tau     = current_node[2];
        fr1_x   = xIndex + current_node[3];
        fr1_y   = yIndex + current_node[4];
        fr2_x   = xIndex + current_node[5];
        fr2_y   = yIndex + current_node[6];

        float ftr1{0};
        float ftr2{0};


        if(ftrID == 0)
        {
            ftr1 = img(fr1_y, fr1_x);
            ftr2 = img(fr2_y, fr2_x);
        }
        else
            for(int i = 0; i < n*n; ++i)
                ftr1 = (float)img(fr1_y + i/n, fr1_x + i%n)*feature[i];

        if((ftr1 - ftr2) <= tau) {
            current_node = tree_nodes.ptr(leftID);
            leftID++;
        }
        else {
            current_node = tree_nodes.ptr(rightID);
            leftID = rightID + 1;
        }
        rightID = current_node[0];
    }

    for (int i = 0; i < lbl_count; ++i)
        layered_hist(yIndex - padding_y, (xIndex - padding_x)*lbl_count + i) = current_node[i+1]/100.0f;
}


void computeLayeredHist_gpu(const cv::Mat& img,
                            cv::Mat_<float>& layered_hist,
                            const cv::Mat_<int>& tree_nodes,
                            const cv::Mat_<float>& features,
                            int lbl_count,
                            int padding_x,
                            int padding_y)
{
    using namespace cv;
    using namespace cv::cuda;

    dim3 block_size(32,32); // 512 threads
    dim3 grid_size;
    grid_size.x = (img.cols + block_size.x - 1)/block_size.x;  // Greater than or equal to image width
    grid_size.y = (img.rows + block_size.y - 1)/block_size.y;  // Greater than or equal to image height

    cv::Mat img_padded = img.clone();
    cv::copyMakeBorder(img_padded, img_padded, padding_y, padding_y, padding_x, padding_x, cv::BORDER_CONSTANT);

    GpuMat img_d(img_padded);
    GpuMat layered_hist_d(layered_hist);
    GpuMat nodes_d(tree_nodes);
    GpuMat features_d(features);


//    compute_layered_hist_gpu<<<grid_size,block_size>>>(img_d, layered_hist_d, nodes_d, features_d, lbl_count, padding_x, padding_y);
    compute_layered_hist_gpu_2b<<<grid_size,block_size>>>(img_d, layered_hist_d, nodes_d, features_d, lbl_count, padding_x, padding_y);
    gpuErrchk( hipPeekAtLastError() );
    hipDeviceSynchronize();
    layered_hist_d.download(layered_hist);
}
